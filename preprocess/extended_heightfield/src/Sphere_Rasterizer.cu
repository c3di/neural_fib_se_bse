#include "hip/hip_runtime.h"
#pragma once

#include "Sphere_Rasterizer.h"

#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>

/* The rasterizer performs z-buffer rasterization by brute force looping all primitives. 
*/
__global__ void rasterize_sphere_kernel(Sphere* spheres,
								        int n_spheres,
								        float2* extended_heightfield, // contains entry/exit information as float2 per pixel
	                                    int2 output_resolution,
										int buffer_length,
										int n_hf_entries,
										float image_plane_z,
										bool debug )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= output_resolution.x)
		return;
	if (idy >= output_resolution.y)
		return;

	int pixel_index = idx * output_resolution.y + idy;

	// initialize extended hf
	for (int i = 0; i < buffer_length; i++)
		extended_heightfield[pixel_index * buffer_length + i] = empty_interval;

	const float pixel_x = (float) idx;
	const float pixel_y = (float) idy;

	int hit_index = 0;

	// loop over all spheres
	for (int sphere_id = 0; sphere_id < n_spheres; sphere_id++)
	{
		const Sphere& sphere = spheres[sphere_id];

		if (debug && idx == 74 && idy == 45)
			printf("  %i : %.2f %.2f %.2f radius %.2f\n", sphere_id, sphere.x, sphere.y, sphere.z, sphere.r);

		const float dz = fabsf( sphere.z - image_plane_z);

		// early termination if sphere behind image plane
		if ( dz <= -sphere.r )
			continue;

		if (debug && idx == 74 && idy == 45)
			printf("    : front of image plance\n");

		// calculate entry and exit point by computing both solutions to r^2 = (x-x0)^2 + (y-y0)^2 + (z-z0)^2
		const float dx = pixel_x - sphere.x;
		const float dy = pixel_y - sphere.y;

		// check if intersection point exists
		if (dx * dx + dy * dy > sphere.r * sphere.r)
			continue;

		const float square_term = sqrtf( sphere.r * sphere.r - dx * dx - dy * dy );
		float entry = sphere.z - square_term;
		float exit  = sphere.z + square_term;

		// handle the case that the sphere is cut by the image place 
		if (entry < image_plane_z)
			entry = image_plane_z;

		if (debug && idx == 74 && idy == 45)
			printf("    : intersect %.2f - %.2f\n", entry, exit);

		extended_heightfield[pixel_index * buffer_length + hit_index] = make_float2( entry, exit );
		hit_index++;

		if (hit_index > buffer_length)
			return;
	}
}

Sphere_Rasterizer::Sphere_Rasterizer(py::array& spheres, std::pair<int, int> output_resolution, int n_hf_entries, int max_buffer_length)
	: output_resolution( make_int2( std::get<0>(output_resolution), std::get<1>(output_resolution) ) )
	, n_hf_entries(n_hf_entries)
{
	allocate_spheres_cpu(spheres);
	if (n_spheres < max_buffer_length)
		buffer_length = n_spheres;
	else
		buffer_length = max_buffer_length;
	presort_spheres();
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);

	extended_heightfield_gpu = allocate_float2_buffer_on_gpu( make_int3( std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length ) );
}

Sphere_Rasterizer::Sphere_Rasterizer(float2* extended_heightfield_gpu, py::array& spheres, std::pair<int, int> output_resolution, int n_hf_entries, int max_buffer_length) 
	: extended_heightfield_gpu(extended_heightfield_gpu)
	, output_resolution( make_int2(std::get<0>(output_resolution), std::get<1>(output_resolution) ) )
	, n_hf_entries(n_hf_entries)
{
	allocate_spheres_cpu(spheres);
	if (n_spheres < max_buffer_length)
		buffer_length = n_spheres;
	else
		buffer_length = max_buffer_length;
	presort_spheres();
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);
}


Sphere_Rasterizer::~Sphere_Rasterizer()
{
	// todo fix leaks
}

py::array_t<float> Sphere_Rasterizer::rasterize_spheres_py( float image_plane )
{
	rasterize_spheres( image_plane );
	auto pyarray = create_py_array(output_resolution.x, output_resolution.y, buffer_length * 2);
	hipMemcpy(pyarray.request().ptr, extended_heightfield_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * buffer_length, hipMemcpyDeviceToHost );
	return pyarray;
}

void Sphere_Rasterizer::rasterize_spheres( float image_plane )
{
	int2 grid_size = output_resolution;
	dim3 block_size(32, 32);
	dim3 num_blocks((grid_size.x + block_size.x - 1) / block_size.x, (grid_size.y + block_size.y - 1) / block_size.y);
	rasterize_sphere_kernel << <num_blocks, block_size >> > (spheres_gpu, spheres_cpu.size(), extended_heightfield_gpu, output_resolution, buffer_length, n_hf_entries, image_plane, false);
	throw_on_cuda_error();
}

void Sphere_Rasterizer::allocate_spheres_cpu(py::array& spheres)
{
	py::buffer_info info = spheres.request();
	if (info.ndim != 2)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	if (info.shape[1] != 4)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	if (info.format != "f")
		throw std::invalid_argument("spheres array is expected to be of dtype float32, found " + info.format);
	n_spheres = info.shape[0];
	spheres_cpu.resize(n_spheres);
	float* ptr = (float*) info.ptr;
	for (size_t i = 0; i < info.shape[0]; i++)
	{
		spheres_cpu[i].x = *(ptr++);
		spheres_cpu[i].y = *(ptr++);
		spheres_cpu[i].z = *(ptr++);
		spheres_cpu[i].r = *(ptr++);
	}
}

Sphere* Sphere_Rasterizer::allocate_spheres_on_gpu( const std::vector<Sphere>& spheres_cpu )
{
	Sphere* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(Sphere) * n_spheres);
	hipMemcpy(ptr_gpu, &spheres_cpu[0], sizeof(Sphere) * n_spheres, hipMemcpyHostToDevice);
	return ptr_gpu;
}

void Sphere_Rasterizer::presort_spheres()
{
	struct {
		bool operator()(Sphere a, Sphere b) const { return a.z + a.r < b.z + b.r; }
	} bottomPosition;
	std::sort( spheres_cpu.begin(), spheres_cpu.end(), bottomPosition );
}
