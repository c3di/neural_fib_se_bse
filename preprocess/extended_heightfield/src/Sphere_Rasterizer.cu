#include "hip/hip_runtime.h"
#pragma once

#include "Sphere_Rasterizer.h"

#include <iostream>
#include <stdexcept>

/* The rasterizer performs z-buffer rasterization by brute force looping all primitives. 
*/

#define empty 65535.0f
#define empty_interval make_float2( empty, empty )

void throw_on_cuda_error()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(error));
	}
};

__global__ void resolve_csg_kernel(
	float2* extended_heightfield,
	int3 output_resolution,
	int n_hf_entries)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= output_resolution.x)
		return;
	if (idx >= output_resolution.y)
		return;

	int pixel_index = idx * output_resolution.y + idy;

	int i = output_resolution.z;
	// while (extended_heightfield[pixel_index * output_resolution.z + i].x != empty) 
	while ( i > 0 )
	{
		for (int j = 0; j < i-1; j++)
		{
			if (idx == 8 && idy == 4)
			{
				for (int ii = 0; ii < output_resolution.z; ii++)
					printf("    %i : %.2f %.2f\n", ii, extended_heightfield[pixel_index * output_resolution.z + ii].x, extended_heightfield[pixel_index * output_resolution.z + ii].y);
			}

			float2& a = extended_heightfield[pixel_index * output_resolution.z + j];
			float2& b = extended_heightfield[pixel_index * output_resolution.z + j + 1];

			// case A: order ok
			if (a.y < b.x)
			{
				if (idx == 8 && idy == 4)
					printf("  %i %i noop\n", i, j );
			}

			// case B: swap
			else if (a.x > b.y)
			{
				float2 tmp = b;
				b = a;
				a = tmp;
				if (idx == 8 && idy == 4)
					printf("  %i %i swap %.2f %.2f <-> %.2f %.2f \n", i, j, a.x, a.y, b.x, b.y );
			}

			// case C: merge
			else {
				float new_x = fminf(a.x, b.x);
				float new_y = fmaxf(a.y, b.y);
				if (idx == 8 && idy == 4) 
				{
					printf("  %i %i merge %.2f %.2f <-> %.2f %.2f \n", i, j, a.x, a.y, b.x, b.y);
				}
				a = make_float2(new_x, new_y);
				b = empty_interval;
				continue;
			} // merge

		}
		i--;
	}
}

__global__ void rasterize_sphere_kernel(Sphere* spheres,
								        int n_spheres,
								        float2* extended_heightfield, // contains entry/exit information as float2 per pixel
	                                    int2 output_resolution,
										int n_hf_entries,
										float image_plane_z )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= output_resolution.x)
		return;
	if (idx >= output_resolution.y)
		return;

	int pixel_index = idx * output_resolution.y + idy;

	// initialize extended hf
	for (int i = 0; i < n_spheres; i++)
		extended_heightfield[pixel_index * n_spheres + i] = empty_interval;

	const float pixel_x = (float) idx;
	const float pixel_y = (float) idy;

	int hit_index = 0;

	// loop over all spheres
	for (int sphere_id = 0; sphere_id < n_spheres; sphere_id++)
	{
		const Sphere& sphere = spheres[sphere_id];

		const float dz = fabsf( sphere.z - image_plane_z);

		// early termination if sphere behind image plane
		if ( dz <= -sphere.r )
			continue;

		// calculate entry and exit point by computing both solutions to r^2 = (x-x0)^2 + (y-y0)^2 + (z-z0)^2
		const float dx = pixel_x - sphere.x;
		const float dy = pixel_y - sphere.y;

		// check if intersection point exists
		if (dx * dx + dy * dy > sphere.r * sphere.r)
			continue;

		const float square_term = sqrtf( sphere.r * sphere.r - dx * dx - dy * dy );
		float entry = sphere.z - square_term;
		float exit  = sphere.z + square_term;

		// handle the case that the sphere is cut by the image place 
		if (entry < image_plane_z)
			entry = image_plane_z;

		if (idx == 8 && idy == 4)
			printf("  creating %.2f %.2f\n", entry, exit );

		extended_heightfield[pixel_index * n_spheres + hit_index] = make_float2( entry, exit );
		hit_index++;
	}
}

/*Sphere_Rasterizer_Kernel::Sphere_Rasterizer_Kernel(std::vector<Sphere>& spheres, int2 output_resolution, int n_hf_entries)
	: spheres_cpu(spheres)
	, output_resolution(output_resolution)
	, n_hf_entries(n_hf_entries)
{
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);
	extended_heightfield_gpu = allocate_extended_heightfield_on_gpu();
} */

Sphere_Rasterizer::Sphere_Rasterizer(py::array& spheres, std::pair<int, int> output_resolution, int n_hf_entries) // , std::pair<int, int> output_resolution, 
	: output_resolution( make_int2( std::get<0>(output_resolution), std::get<1>(output_resolution) ) )
	, n_hf_entries(n_hf_entries)
{
	std::cout << "creating extended heightfield of resolution " << std::get<0>(output_resolution) << "/" << std::get<1>(output_resolution) << std::endl;
	allocate_spheres_cpu(spheres);
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);
	extended_heightfield_gpu = allocate_extended_heightfield_on_gpu();
}

Sphere_Rasterizer::~Sphere_Rasterizer()
{
}

py::array_t<float> Sphere_Rasterizer::rasterize_spheres_py( float image_plane )
{
	rasterize_spheres( image_plane );
	auto pyarray = create_py_array(output_resolution.x, output_resolution.y, n_spheres * 2);
	hipMemcpy(pyarray.request().ptr, extended_heightfield_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * n_spheres, hipMemcpyDeviceToHost );
	return pyarray;
}

void Sphere_Rasterizer::rasterize_spheres( float image_plane )
{
	int2 grid_size = output_resolution;
	dim3 block_size(32, 32);
	dim3 num_blocks((grid_size.x + block_size.x - 1) / block_size.x, (grid_size.y + block_size.y - 1) / block_size.y);
	rasterize_sphere_kernel << <num_blocks, block_size >> > (spheres_gpu, spheres_cpu.size(), extended_heightfield_gpu, output_resolution, n_hf_entries, image_plane);

	int3 buffer_size = make_int3(output_resolution.x, output_resolution.y, n_spheres);
	resolve_csg_kernel << <num_blocks, block_size >> > (extended_heightfield_gpu, buffer_size, n_hf_entries);
	throw_on_cuda_error();
}

py::array_t<float> Sphere_Rasterizer::create_py_array( int shape0, int shape1, int shape2 )
{
	return py::array(py::buffer_info(
			         nullptr,                                                                   /* Pointer to data (nullptr -> ask NumPy to allocate!) */
                     sizeof(float),                                                              /* Size of one item */
                     py::format_descriptor<float>::value,                                        /* Buffer format */
                     3,																		     /* How many dimensions? */
                     { shape0, shape1, shape2 },                                                 /* Number of elements for each dimension */
                     { shape1 * shape2 * sizeof(float), shape2 * sizeof(float), sizeof(float) }  /* Strides for each dimension */
	));
}

void Sphere_Rasterizer::allocate_spheres_cpu(py::array& spheres)
{
py::buffer_info info = spheres.request();
	if (info.ndim != 2)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	if (info.shape[1] != 4)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	n_spheres = info.shape[0];
	spheres_cpu.resize(n_spheres);
	double* ptr = (double*) info.ptr;
	for (size_t i = 0; i < info.shape[0]; i++)
	{
		spheres_cpu[i].x = *(ptr++);
		spheres_cpu[i].y = *(ptr++);
		spheres_cpu[i].z = *(ptr++);
		spheres_cpu[i].r = *(ptr++);
	}
	for (auto it : spheres_cpu)
		std::cout << "Sphere: " << it.x << " " << it.y << " " << it.z << " / " << it.r << std::endl;
}

Sphere* Sphere_Rasterizer::allocate_spheres_on_gpu( const std::vector<Sphere>& spheres_cpu )
{
	int n_spheres = (int) spheres_cpu.size();
	Sphere* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(Sphere) * n_spheres);
	hipMemcpy(ptr_gpu, &spheres_cpu[0], sizeof(Sphere) * n_spheres, hipMemcpyHostToDevice);
	return ptr_gpu;
}

float2* Sphere_Rasterizer::allocate_extended_heightfield_on_gpu()
{
	float2* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * n_spheres);
	return ptr_gpu;
}
