#include "hip/hip_runtime.h"
#pragma once

#include "Sphere_Rasterizer_Kernel.h"

#include <iostream>
#include <stdexcept>

/* The rasterizer performs z-buffer rasterization by brute force looping all primitives. 
*/

#define empty 65535.0f
#define empty_interval make_float2( empty, empty )

void throw_on_cuda_error()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(error));
	}
};

__global__ void rasterize_sphere_kernel(Sphere* spheres,
								        int n_spheres,
								        float2* extended_heightfield,
	                                    int2 output_resolution,
										int n_hf_entries,
										float image_plane_z )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= output_resolution.x)
		return;
	if (idx >= output_resolution.y)
		return;

	int pixel_index = idx * output_resolution.y + idy;

	// initialize extended hf
	for (int i = 0; i < n_hf_entries; i++)
		extended_heightfield[pixel_index * n_hf_entries + i] = empty_interval;

	const float pixel_x = (float) idx;
	const float pixel_y = (float) idy;
	// loop over all spheres
	for (int sphere_id = 0; sphere_id < n_spheres; sphere_id++)
	{
		const Sphere& sphere = spheres[sphere_id];
		const float dz = fabsf( sphere.z - image_plane_z);
		if ( dz >= sphere.r )
			continue;

		// early termination if the pixel coordinates are outside the sphere
		const float r = sqrtf( sphere.r * sphere.r - dz * dz );
		if ( pixel_x < sphere.x - r || pixel_x > sphere.x + r )
			continue;
		if ( pixel_y < sphere.x - r || pixel_x > sphere.x + r )
			continue;

		// calculate entry and exit point by computing both solutions to r^2 = (x-x0)^2 + (y-y0)^2 + (z-z0)^2
		const float dx = pixel_x - sphere.x;
		const float dy = pixel_y - sphere.y;

		const float square_term = sqrtf( sphere.r * sphere.r - dx * dx - dy * dy );
		float entry = sphere.z - square_term;
		float exit  = sphere.z + square_term;

		// handle the case that the sphere is cut by the image place 
		if (entry < image_plane_z)
			entry = image_plane_z;

		unsigned int i = 0;
		while (extended_heightfield[i].x < entry)
			i++;
		float2 replaced_number = extended_heightfield[i];
		extended_heightfield[i] = make_float2( entry, exit );
		i++;
		do
		{
			float2 tmp = extended_heightfield[i];
			extended_heightfield[i] = replaced_number;
			replaced_number = tmp;
		} while (replaced_number.x < empty);

	}
}

/*Sphere_Rasterizer_Kernel::Sphere_Rasterizer_Kernel(std::vector<Sphere>& spheres, int2 output_resolution, int n_hf_entries)
	: spheres_cpu(spheres)
	, output_resolution(output_resolution)
	, n_hf_entries(n_hf_entries)
{
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);
	extended_heightfield_gpu = allocate_extended_heightfield_on_gpu();
} */

Sphere_Rasterizer_Kernel::Sphere_Rasterizer_Kernel(py::array& spheres, std::pair<int, int> output_resolution, int n_hf_entries) // , std::pair<int, int> output_resolution, 
	// : output_resolution( make_int2( std::get<0>(output_resolution), std::get<1>(output_resolution) ) )
	// , n_hf_entries(n_hf_entries)
{
	std::cout << "creating extended heightfield of resolution " << std::get<0>(output_resolution) << "/" << std::get<1>(output_resolution) << std::endl;
	allocate_spheres_cpu(spheres);
	spheres_gpu = allocate_spheres_on_gpu(spheres_cpu);
	extended_heightfield_gpu = allocate_extended_heightfield_on_gpu();
}

Sphere_Rasterizer_Kernel::~Sphere_Rasterizer_Kernel()
{
}

std::vector<std::tuple<float, float>> Sphere_Rasterizer_Kernel::rasterize_spheres( float image_plane )
{
	return extended_heightfield_cpu;
}

void Sphere_Rasterizer_Kernel::allocate_spheres_cpu(py::array& spheres)
{
	py::buffer_info info = spheres.request();
	if (info.ndim != 2)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	if (info.shape[1] != 4)
		throw std::invalid_argument("spheres array is expected to be of dimensions nx4");
	spheres_cpu.resize(info.shape[0]);
	double* ptr = (double*) info.ptr;
	for (size_t i = 0; i < info.shape[0]; i++)
	{
		spheres_cpu[i].x = *(ptr++);
		spheres_cpu[i].y = *(ptr++);
		spheres_cpu[i].z = *(ptr++);
		spheres_cpu[i].r = *(ptr++);
	}
	for (auto it : spheres_cpu)
		std::cout << "Sphere: " << it.x << " " << it.y << " " << it.z << " / " << it.r << std::endl;
}

void Sphere_Rasterizer_Kernel::call_kernel()
{
	int2 grid_size = output_resolution;
	dim3 block_size(32, 32);
	dim3 num_blocks((grid_size.x + block_size.x - 1) / block_size.x, (grid_size.y + block_size.y - 1) / block_size.y );
	rasterize_sphere_kernel << <num_blocks, block_size >> > (spheres_gpu, spheres_cpu.size(), extended_heightfield_gpu, output_resolution, n_hf_entries, image_plane);
	throw_on_cuda_error();
}

Sphere* Sphere_Rasterizer_Kernel::allocate_spheres_on_gpu( const std::vector<Sphere>& spheres_cpu )
{
	int n_spheres = (int) spheres_cpu.size();
	Sphere* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(Sphere) * n_spheres);
	hipMemcpy(ptr_gpu, &spheres_cpu[0], sizeof(Sphere) * n_spheres, hipMemcpyHostToDevice);
	return ptr_gpu;
}

float2* Sphere_Rasterizer_Kernel::allocate_extended_heightfield_on_gpu()
{
	float2* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * n_hf_entries);
	return ptr_gpu;
}