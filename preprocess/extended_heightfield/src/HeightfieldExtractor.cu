#include "hip/hip_runtime.h"
#pragma once

#include "HeightFieldExtractor.h"
#include "Sphere_Rasterizer.h"
#include "CSG_Resolver.h"

#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>

__global__ void collect_result_kernel( float2* extended_heightfield, float2* result, int3 output_resolution, int n_hf_entries )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx >= output_resolution.x)
		return;
	if (idy >= output_resolution.y)
		return;

	int pixel_index = idx * output_resolution.y + idy;
	for (int i = 0; i < n_hf_entries; i++)
	{
		result[pixel_index * n_hf_entries + i] = extended_heightfield[pixel_index * output_resolution.z + i];
	}
}

HeightFieldExtractor::HeightFieldExtractor(py::array& spheres, std::pair<int, int> output_resolution, int n_hf_entries, int max_buffer_length )
	: output_resolution( make_int2(std::get<0>(output_resolution), std::get<1>(output_resolution)) )
	, n_hf_entries(n_hf_entries)
	, buffer_length(max_buffer_length)
{
	extended_heightfield_gpu = allocate_float2_buffer_on_gpu(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length));
	result_gpu = allocate_float2_buffer_on_gpu(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), n_hf_entries));
	sphere_rasterizer = new Sphere_Rasterizer(extended_heightfield_gpu, spheres, output_resolution, n_hf_entries, max_buffer_length);
	csg_resolver = new CSG_Resolver(extended_heightfield_gpu, make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length), n_hf_entries );
}

HeightFieldExtractor::HeightFieldExtractor(std::vector<Sphere>& spheres, std::pair<int, int> output_resolution, int n_hf_entries, int max_buffer_length )
	: output_resolution(make_int2(std::get<0>(output_resolution), std::get<1>(output_resolution)))
	, n_hf_entries(n_hf_entries)
	, buffer_length(max_buffer_length)
{
	extended_heightfield_gpu = allocate_float2_buffer_on_gpu(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length));
	result_gpu = allocate_float2_buffer_on_gpu(make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), n_hf_entries));
	sphere_rasterizer = new Sphere_Rasterizer(extended_heightfield_gpu, spheres, output_resolution, n_hf_entries, max_buffer_length);
	csg_resolver = new CSG_Resolver(extended_heightfield_gpu, make_int3(std::get<0>(output_resolution), std::get<1>(output_resolution), buffer_length), n_hf_entries);
}

HeightFieldExtractor::~HeightFieldExtractor()
{
	delete csg_resolver;
	// delete sphere_rasterizer;
	// delete result_gpu;
}

std::pair<std::vector<float>, std::vector<float>> HeightFieldExtractor::extract_data_representation(float image_plane)
{
	rasterize(image_plane);
	return std::pair<std::vector<float>, std::vector<float>>(collect_extended_heightfield(), sphere_rasterizer->get_normal_map());
}

std::pair< py::array_t<float>, py::array_t<float>>  HeightFieldExtractor::extract_data_representation_py(float image_plane)
{
	rasterize( image_plane );
	return std::pair< py::array_t<float>, py::array_t<float>>( collect_extended_heightfield_py(), sphere_rasterizer->get_normal_map_py() );
}

void HeightFieldExtractor::rasterize(float image_plane)
{
	sphere_rasterizer->rasterize_spheres(image_plane);
	csg_resolver->resolve_csg(image_plane);
}

py::array_t<float> HeightFieldExtractor::collect_extended_heightfield_py()
{
	call_result_collection_kernel();
	auto pyarray = create_py_array(output_resolution.x, output_resolution.y, n_hf_entries * 2);
	hipMemcpy(pyarray.request().ptr, result_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * n_hf_entries, hipMemcpyDeviceToHost);
	return pyarray;
}

std::vector<float> HeightFieldExtractor::collect_extended_heightfield()
{
	call_result_collection_kernel();
	std::vector<float> result_cpu(2 * output_resolution.x * output_resolution.y * n_hf_entries);
	hipMemcpy(&result_cpu[0], result_gpu, sizeof(float2) * output_resolution.x * output_resolution.y * n_hf_entries, hipMemcpyDeviceToHost);
	return result_cpu;
}	

void HeightFieldExtractor::call_result_collection_kernel()
{
	dim3 block_size(32, 32);
	dim3 num_blocks((output_resolution.x + block_size.x - 1) / block_size.x, (output_resolution.y + block_size.y - 1) / block_size.y);
	int3 buffer_size = make_int3(output_resolution.x, output_resolution.y, buffer_length);
	collect_result_kernel << <num_blocks, block_size >> > (extended_heightfield_gpu, result_gpu, buffer_size, n_hf_entries);
}
