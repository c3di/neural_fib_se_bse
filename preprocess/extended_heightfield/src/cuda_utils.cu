#include "hip/hip_runtime.h"
#pragma once

#include <stdexcept>

#include "cuda_utils.h"

void throw_on_cuda_error()
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		throw std::runtime_error(hipGetErrorString(error));
	}
};

float2* allocate_float2_buffer_on_gpu( int3 buffer_size )
{
	float2* ptr_gpu;
	hipMalloc((void**)&ptr_gpu, sizeof(float2) * buffer_size.x * buffer_size.y * buffer_size.z );
	return ptr_gpu;
}